

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <limits.h>
#include <float.h>
#include <math.h>
#include <sys/time.h>

#include <pthread.h>

#define NTHREADS      8
#define ARRAYSIZE   5120
#define ITERATIONS   ARRAYSIZE / NTHREADS

float feature_vect[] = { 2.240018, 2.2570236, 0.11304555, -0.21307051, 0.8988138, 0.039065503, 0.023874786, 0.13153112, 0.15324382, 0.16986738, -0.020297153, -0.26773554, 0.40202165, 0.35923952,
			0.060746543, 0.35402644, 0.086052455, -0.10499257, 0.04395058, 0.026407119, -0.48301497, 0.120889395, 0.67980754, -0.19875681, -0.5443737, -0.039534688, 0.20888293, 0.054865785, -0.4846478, 0.1, 0.1, 0.1};

float *means_vect;
float *precs_vect;
float *weight_vect;
float *factor_vect;

float *score_vect;
float *cpu_score_vect;
float *pthread_score_vect;

//pthread_mutex_t sum_mutex;


__device__ __constant__ float logZero = -3.4028235E38;

//__device__ __constant__ float logBase = 1.0001;

__device__ __constant__ float maxLogValue = 7097004.5;
__device__ __constant__ float minLogValue = -7443538.0;

__device__ __constant__ float naturalLogBase = (float) 1.00011595E-4;
__device__ __constant__ float inverseNaturalLogBase = 9998.841;

// fixed for a given accoustic model
__device__ __constant__  int comp_size = 32;
__device__ __constant__  int feat_size = 29;
__device__ __constant__  int senone_size = 5120;

extern "C"

__global__ void computeScore(const float *feature_vect, float *means_vect, float *precs_vect, float *weight_vect, float *factor_vect, float *score_vect)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	//  while (i < senone_size) {
	if (i < senone_size) {

		float local_score_vect = logZero;
		//        int senone_id = senone_vect[i];

		#pragma unroll 32
		for (int j = 0; j < comp_size; j++) {
			// getScore
			float logDval = 0.0f;
			#pragma unroll 29
			for (int k = 0; k < feat_size; k++) {
		 	  //  int idx = k + feat_size*j + i*comp_size*feat_size;
		 	    int idx = i + senone_size*j + k*comp_size*senone_size;
				//int idx = i + senone_size*j + k*comp_size*senone_size;
				float logDiff = feature_vect[k] - means_vect[idx];
				logDval += logDiff * logDiff * precs_vect[idx];
			}

			// Convert to the appropriate base.
			//logDval = logMath.lnToLog(logDval);
			if (logDval != logZero) {
				logDval = logDval * inverseNaturalLogBase;
			}

			int idx2 = i + j*senone_size;
		//	int idx2 = j + i*comp_size;

			//    int idx2 = senone_id + j*comp_size;

			// Add the precomputed factor, with the appropriate sign.
			//  logDval -= mixtureComponents[i].getLogPreComputedGaussianFactor();
			logDval -= factor_vect[idx2];

			/*      if (Float.isNaN(logDval)) {
                System.out.println("gs is Nan, converting to 0");
                logDval = logZero;
            }*/

			if (logDval < logZero) {
				logDval = logZero;
			}
			// end of getScore

			//      float logVal2 = logDval + logMixtureWeights[i];
			float logVal2 = logDval + weight_vect[idx2];

			//float logVal2 = mixtureComponents[i].getScore(featureVector) + logMixtureWeights[i];
			float logHighestValue = local_score_vect;
			float logDifference = local_score_vect - logVal2;

			// difference is always a positive number
			//            float logHighestValue = (logDifference1 < 0)? logVal2 : local_score_vect;
			//           float logDifference =  (logDifference1 < 0)? -logDifference1 : logDifference1;
			if (logDifference < 0) {
				logHighestValue = logVal2;
				logDifference = -logDifference;
			}

			//double logInnerSummation = logToLinear(-logDifference);
			float logValue = -logDifference;
			float logInnerSummation;
			if (logValue < minLogValue) {
				logInnerSummation = 0.0;
			} else if (logValue > maxLogValue) {
				logInnerSummation = FLT_MAX;
			} else {
				if (logValue == logZero) {
					logValue = logZero;
				} else {
					logValue = logValue * naturalLogBase;
				}
				logInnerSummation = __expf(logValue);
			}

			logInnerSummation += 1.0;

			//float actual_comp = linearToLog(logInnerSummation);
			float returnLogValue;
			if (logInnerSummation <= 0.0) {
				returnLogValue = logZero;
			} else {
				returnLogValue = __logf(logInnerSummation) * inverseNaturalLogBase;
				if (returnLogValue > FLT_MAX) {
					returnLogValue = FLT_MAX;
				} else if (returnLogValue < -FLT_MAX) {
					returnLogValue = -FLT_MAX;
				}
			}
			// sum log
			local_score_vect = logHighestValue + returnLogValue;
		}

		score_vect[i] = local_score_vect;

		//        i += blockDim.x * gridDim.x;

	}
	// __syncthreads();

}


void computeScore_seq(float* feature_vect, float* means_vect, float * precs_vect, float* weight_vect, float* factor_vect, float * score_vect) {

	float logZero = -3.4028235E38;

	//float logBase = 1.0001;

	float maxLogValue = 7097004.5;
	float minLogValue = -7443538.0;

	float naturalLogBase = (float) 1.00011595E-4;
	float inverseNaturalLogBase = 9998.841;

	int comp_size = 32;
	int feat_size = 29;
	int senone_size = 5120;

	for (int i = 0; i < senone_size; i++) {

		score_vect[i] = logZero;
		//            int sen_id = senone_ids[i];

		for (int j = 0; j < comp_size; j++) {

			// getScore
			// idx = k + D*j + i*W*D
			float logDval = 0.0f;
			for (int k = 0; k < feat_size; k++) {
				//float logDiff = featureVector[k] - mean_trans[k];
				//logDval += logDiff * logDiff * prec_trans[k];
				int idx = k + comp_size*j + i*comp_size*comp_size;
				float logDiff = feature_vect[k] - means_vect[idx];
				logDval += logDiff * logDiff * precs_vect[idx];
			}
			// System.out.println("NEW comp: " + i + " logDval:"+logDval + " after feature_vect");


			// Convert to the appropriate base.
			//logDval = logMath.lnToLog(logDval);
			if (logDval != logZero) {
				logDval = logDval * inverseNaturalLogBase;
			}

			int idx2 = j + i*comp_size;

			// Add the precomputed factor, with the appropriate sign.
			//  logDval -= mixtureComponents[i].getLogPreComputedGaussianFactor();
			logDval -= factor_vect[idx2];

			/*      if (Float.isNaN(logDval)) {
                    System.out.println("gs is Nan, converting to 0");
                    logDval = logZero;
                }*/

			if (logDval < logZero) {
				logDval = logZero;
			}
			// end of getScore
			//          System.out.println("NEW comp: " + i + " logDval:"+logDval + " after getScore, preFactor_array");

			//      float logVal2 = logDval + logMixtureWeights[i];
			float logVal2 = logDval + weight_vect[idx2];
			//        System.out.println("NEW comp: " + i + " logVal2:"+logVal2 + " after mixWeightArray");

			//float logVal2 = mixtureComponents[i].getScore(featureVector) + logMixtureWeights[i];
			float logHighestValue = score_vect[i];
			float logDifference = score_vect[i] - logVal2;

			// difference is always a positive number
			if (logDifference < 0) {
				logHighestValue = logVal2;
				logDifference = -logDifference;
			}

			//double logInnerSummation = logToLinear(-logDifference);
			float logValue = -logDifference;
			float logInnerSummation;
			if (logValue < minLogValue) {
				logInnerSummation = 0.0;
			} else if (logValue > maxLogValue) {
				logInnerSummation = FLT_MAX;

			} else {
				if (logValue == logZero) {
					logValue = logZero;
				} else {
					logValue = logValue * naturalLogBase;
				}
				logInnerSummation = exp(logValue);
			}

			logInnerSummation += 1.0;

			//float actual_comp = linearToLog(logInnerSummation);
			float returnLogValue;
			if (logInnerSummation <= 0.0) {
				returnLogValue = logZero;

			} else {
				returnLogValue = (float) (log(logInnerSummation) * inverseNaturalLogBase);
				if (returnLogValue > FLT_MAX) {
					returnLogValue = FLT_MAX;
				} else if (returnLogValue < -FLT_MAX) {
					returnLogValue = -FLT_MAX;
				}
			}
			// sum log
			score_vect[i] = logHighestValue + returnLogValue;
		}
	}
	//    }
}

void *computeScore_thread(void *tid)
//void *computeScore_thread(void *tid, float* feature_vect, float* means_vect, float * precs_vect, float* weight_vect, float* factor_vect, float * score_vect)
{
	float logZero = -3.4028235E38;

	//float logBase = 1.0001;

	float maxLogValue = 7097004.5;
	float minLogValue = -7443538.0;

	float naturalLogBase = (float) 1.00011595E-4;
	float inverseNaturalLogBase = 9998.841;

	int comp_size = 32;
	int feat_size = 29;
	int senone_size = 5120;

	int i, start, *mytid, end;

	mytid = (int *) tid;
	start = (*mytid * ITERATIONS);
	end = start + ITERATIONS;
	printf ("Thread %d doing iterations %d to %d\n",*mytid,start,end-1);

	//for (int i = 0; i < senone_size; i++) {
	 for (i=start; i < end ; i++) {

		pthread_score_vect[i] = logZero;
		//            int sen_id = senone_ids[i];

		for (int j = 0; j < comp_size; j++) {

			// getScore
			// idx = k + D*j + i*W*D
			float logDval = 0.0f;
			for (int k = 0; k < feat_size; k++) {
				//float logDiff = featureVector[k] - mean_trans[k];
				//logDval += logDiff * logDiff * prec_trans[k];
				int idx = k + comp_size*j + i*comp_size*comp_size;
				float logDiff = feature_vect[k] - means_vect[idx];
				logDval += logDiff * logDiff * precs_vect[idx];
			}
			// System.out.println("NEW comp: " + i + " logDval:"+logDval + " after feature_vect");


			// Convert to the appropriate base.
			//logDval = logMath.lnToLog(logDval);
			if (logDval != logZero) {
				logDval = logDval * inverseNaturalLogBase;
			}

			int idx2 = j + i*comp_size;

			// Add the precomputed factor, with the appropriate sign.
			//  logDval -= mixtureComponents[i].getLogPreComputedGaussianFactor();
			logDval -= factor_vect[idx2];

			/*      if (Float.isNaN(logDval)) {
                    System.out.println("gs is Nan, converting to 0");
                    logDval = logZero;
                }*/

			if (logDval < logZero) {
				logDval = logZero;
			}
			// end of getScore
			//          System.out.println("NEW comp: " + i + " logDval:"+logDval + " after getScore, preFactor_array");

			//      float logVal2 = logDval + logMixtureWeights[i];
			float logVal2 = logDval + weight_vect[idx2];
			//        System.out.println("NEW comp: " + i + " logVal2:"+logVal2 + " after mixWeightArray");

			//float logVal2 = mixtureComponents[i].getScore(featureVector) + logMixtureWeights[i];
			float logHighestValue = score_vect[i];
			float logDifference = score_vect[i] - logVal2;

			// difference is always a positive number
			if (logDifference < 0) {
				logHighestValue = logVal2;
				logDifference = -logDifference;
			}

			//double logInnerSummation = logToLinear(-logDifference);
			float logValue = -logDifference;
			float logInnerSummation;
			if (logValue < minLogValue) {
				logInnerSummation = 0.0;
			} else if (logValue > maxLogValue) {
				logInnerSummation = FLT_MAX;

			} else {
				if (logValue == logZero) {
					logValue = logZero;
				} else {
					logValue = logValue * naturalLogBase;
				}
				logInnerSummation = exp(logValue);
			}

			logInnerSummation += 1.0;

			//float actual_comp = linearToLog(logInnerSummation);
			float returnLogValue;
			if (logInnerSummation <= 0.0) {
				returnLogValue = logZero;

			} else {
				returnLogValue = (float) (log(logInnerSummation) * inverseNaturalLogBase);
				if (returnLogValue > FLT_MAX) {
					returnLogValue = FLT_MAX;
				} else if (returnLogValue < -FLT_MAX) {
					returnLogValue = -FLT_MAX;
				}
			}
			// sum log
			pthread_score_vect[i] = logHighestValue + returnLogValue;
		}
	}

	  pthread_exit(NULL);
	//    }
}

float calculateMiliseconds(timeval t1,timeval t2) {
	float elapsedTime;
	elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;
	elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;
	return elapsedTime;
}


int main()
{


	float *dev_feat_vect;

	timeval t1,t2;
	float cuda_elapsedTime;
	float cpu_elapsedTime;
	float par_elapsedTime;
	hipEvent_t eStart,eStop;

	int comp_size = 32;
	int senone_size = 5120;

	int means_array_size = senone_size*comp_size*comp_size;
	int comp_array_size = senone_size*comp_size;

	means_vect = (float *)malloc(means_array_size * sizeof(float));
	precs_vect = (float *)malloc(means_array_size * sizeof(float));
	weight_vect = (float *)malloc(comp_array_size * sizeof(float));
	factor_vect = (float *)malloc(comp_array_size * sizeof(float));

	float *means_vect2 = (float *)malloc(means_array_size * sizeof(float));
	float *precs_vect2 = (float *)malloc(means_array_size * sizeof(float));
	float *weight_vect2 = (float *)malloc(comp_array_size * sizeof(float));
	float *factor_vect2 = (float *)malloc(comp_array_size * sizeof(float));

	float *dev_means_vect;
	float *dev_precs_vect;
	float *dev_weight_vect;
	float *dev_factor_vect;

	score_vect = (float *)malloc(senone_size * sizeof(float));
	cpu_score_vect = (float *)malloc(senone_size * sizeof(float));
	pthread_score_vect = (float *)malloc(senone_size * sizeof(float));

	float *dev_score_vect;

	int blockSizeX = 256;
	int gridSizeX = (int) ceil(senone_size / blockSizeX);

	int div_grid = ((int) (gridSizeX / 32));
	gridSizeX = (div_grid+1) * 32 ;
	//  if (gridSizeX < 128)
	//   gridSizeX = 128;


	  int i, start, tids[NTHREADS];
	  pthread_t threads[NTHREADS];
	  pthread_attr_t attr;


	// load model from file
	FILE *fp = fopen("/home/gpuser/cuda/gmm_data.txt", "r");
	if (fp == NULL) { //checks for the file
		printf("\n Can’t open file");
		exit(-1);
	}

	int idx = 0;
	for (int i = 0; i < senone_size; i++) {
		for (int j = 0; j < comp_size; j++) {
			for (int k = 0; k < comp_size; k++) {
				float elem;
				fscanf(fp, "%f", &elem);
				means_vect[idx] = elem;
				idx = idx + 1;
			}
		}
	}

	idx = 0;
	for (int i = 0; i < senone_size; i++) {
		for (int j = 0; j < comp_size; j++) {
			for (int k = 0; k < comp_size; k++) {
				float elem;
				fscanf(fp, "%f", &elem);
				precs_vect[idx] = elem;
				idx = idx + 1;
			}
		}
	}

	idx = 0;
	for (int i = 0; i < senone_size; i++) {
		for (int j = 0; j < comp_size; j++) {
			float elem;
			fscanf(fp, "%f", &elem);
			weight_vect[idx] = elem;
			idx = idx + 1;
		}
	}

	idx = 0;
	for (int i = 0; i < senone_size; i++) {
		for (int j = 0; j < comp_size; j++) {
			float elem;
			fscanf(fp, "%f", &elem);
			factor_vect[idx] = elem;
			idx = idx + 1;
		}
	}

	fclose(fp);

	int idx3 = 0;
	for (int j = 0; j < comp_size; j++) {
		for (int i = 0; i < senone_size; i++) {
			int ij = j + i*comp_size;
			weight_vect2[idx3] = weight_vect[ij];
			factor_vect2[idx3] = factor_vect[ij];
			idx3 += 1;
		}
	}

	int idx4 = 0;
	for (int k = 0; k < comp_size; k++) {
		for (int j = 0; j < comp_size; j++) {
			for (int i = 0; i < senone_size; i++) {
				int ijk = k + comp_size*j + i*comp_size*comp_size;
				means_vect2[idx4] = means_vect[ijk];
				precs_vect2[idx4] = precs_vect[ijk];
				idx4 += 1;
			}
		}
	}

	for (int i = 0; i < senone_size; i++) {
		for (int j = 0; j < comp_size; j++) {
			for (int k = 0; k < 29; k++) {
				int ijk = k + comp_size*j + i*comp_size*comp_size;
				int kji = i + senone_size*j + k*comp_size*senone_size;
				if (means_vect2[kji] != means_vect[ijk]) {
					printf("%f != %f\n", means_vect2[kji], means_vect[ijk]);
				}
			}
		}
	}

	hipEventCreate(&eStart);
	hipEventCreate(&eStop);


	// just one time to load acoustic model
	hipMalloc((void**)&dev_means_vect, sizeof(float)*means_array_size);
	hipMalloc((void**)&dev_precs_vect, sizeof(float)*means_array_size);
	hipMalloc((void**)&dev_weight_vect, sizeof(float)*comp_array_size);
	hipMalloc((void**)&dev_factor_vect, sizeof(float)*comp_array_size);

	hipMemcpy(dev_means_vect, means_vect2, sizeof(float)*means_array_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_precs_vect, precs_vect2, sizeof(float)*means_array_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_weight_vect, weight_vect2, sizeof(float)*comp_array_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_factor_vect, factor_vect2, sizeof(float)*comp_array_size, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_feat_vect, sizeof(float)*comp_size);
	hipMalloc((void**)&dev_score_vect, sizeof(float)*senone_size);


	printf("blockSizeX = %d\n", blockSizeX);
	printf("gridSizeX = %d\n", gridSizeX);


	dim3 block(128);
	dim3 grid;
	grid.x = (senone_size + block.x - 1)/block.x;

	if (grid.x < 32)
		grid.x = 32;

	printf("grid.x = %d\n", grid.x);

	//  gettimeofday(&t1, NULL);

	hipEventRecord(eStart,0);

	// each time needed for computing score of a given feature vect
	hipMemcpy(dev_feat_vect, feature_vect, comp_size*sizeof(float), hipMemcpyHostToDevice);
	//  cudaMemcpy(dev_score_vect, score_vect, senone_size * sizeof(float), cudaMemcpyHostToDevice);


	//void computeScore(const float *feature_vect, float *means_vect, float *precs_vect, float *weight_vect, float *factor_vect, float *score_vect)
	//  computeScore<<<gridSizeX,blockSizeX>>>(dev_feat_vect, dev_means_vect, dev_precs_vect, dev_weight_vect, dev_factor_vect, dev_score_vect);
	computeScore<<<grid,block>>>(dev_feat_vect, dev_means_vect, dev_precs_vect, dev_weight_vect, dev_factor_vect, dev_score_vect);
	//  computeScore<<<128,128>>>(dev_feat_vect, dev_means_vect, dev_precs_vect, dev_weight_vect, dev_factor_vect, dev_score_vect);


	hipMemcpy(score_vect, dev_score_vect, senone_size * sizeof(float), hipMemcpyDeviceToHost);

	//gettimeofday(&t2, NULL);

	hipEventRecord(eStop,0);
	hipEventSynchronize(eStop);

	hipEventElapsedTime(&cuda_elapsedTime,eStart,eStop);
	//cuda_elapsedTime = calculateMiliseconds(t1,t2);
	printf("CUDA Time=%4.3f ms\n", cuda_elapsedTime);


	// CPU side

	gettimeofday(&t1, NULL);
	computeScore_seq(feature_vect, means_vect, precs_vect, weight_vect, factor_vect, cpu_score_vect);
	gettimeofday(&t2, NULL);

	cpu_elapsedTime = calculateMiliseconds(t1,t2);
	printf("\nCPU Time=%4.3f ms\n",  cpu_elapsedTime);


	for (int k = 0; k < senone_size; k++) {
		if (abs(abs(cpu_score_vect[k] - score_vect[k]) / cpu_score_vect[k]) > 0.01) {
			printf("ERROR on computing scores: CPU %.3f != GPU %.3f\n", cpu_score_vect[k], score_vect[k]);
			//	printf(abs(cpu_score_vect[k] - score_vect[k]) / cpu_score_vect[k]);
		}
	}



	gettimeofday(&t1, NULL);

	  pthread_attr_init(&attr);
	  pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_JOINABLE);
	  for (i=0; i<NTHREADS; i++) {
	    tids[i] = i;
	    pthread_create(&threads[i], &attr, computeScore_thread, (void *) &tids[i]);
	  }

//	  printf ("Waiting for threads to finish.");
	  for (i=0; i<NTHREADS; i++) {
	    pthread_join(threads[i], NULL);
	  }
	//  printf("Done.");

      gettimeofday(&t2, NULL);

      par_elapsedTime = calculateMiliseconds(t1,t2);
	  printf("\nCPU Par Time=%4.3f ms\n",  par_elapsedTime);


	printf("\nCPU Par speedup over CPU = %4.3f\n",  cpu_elapsedTime/par_elapsedTime);

	printf("\nGPU speedup over CPU = %4.3f\n",  cpu_elapsedTime/cuda_elapsedTime);

	printf("\nGPU speedup over CPU Par = %4.3f\n",  par_elapsedTime/cuda_elapsedTime);

	free(means_vect);
	free(precs_vect);

	free(weight_vect);
	free(factor_vect);

	free(score_vect);
	free(cpu_score_vect);
	free(pthread_score_vect);

	hipFree(dev_means_vect);
	hipFree(dev_precs_vect);
	hipFree(dev_weight_vect);
	hipFree(dev_factor_vect);

	hipFree(dev_feat_vect);
	hipFree(dev_score_vect);

	  /* Clean up and exit */
	  pthread_attr_destroy(&attr);
	  //pthread_exit (NULL);

}
