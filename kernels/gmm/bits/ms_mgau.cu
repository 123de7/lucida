#include "hip/hip_runtime.h"
/* -*- c-basic-offset: 4; indent-tabs-mode: nil -*- */
/* ====================================================================
 * Copyright (c) 1999-2004 Carnegie Mellon University.  All rights
 * reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer. 
 *
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in
 *    the documentation and/or other materials provided with the
 *    distribution.
 *
 * This work was supported in part by funding from the Defense Advanced 
 * Research Projects Agency and the National Science Foundation of the 
 * United States of America, and the CMU Sphinx Speech Consortium.
 *
 * THIS SOFTWARE IS PROVIDED BY CARNEGIE MELLON UNIVERSITY ``AS IS'' AND 
 * ANY EXPRESSED OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, 
 * THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL CARNEGIE MELLON UNIVERSITY
 * NOR ITS EMPLOYEES BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, 
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY 
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT 
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE 
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * ====================================================================
 *
 */
/*
 * ms_mgau.c -- Essentially a wrapper that wrap up gauden and
 * senone. It supports multi-stream. 
 *
 *
 * **********************************************
 * CMU ARPA Speech Project
 *
 * Copyright (c) 1997 Carnegie Mellon University.
 * ALL RIGHTS RESERVED.
 * **********************************************
 * HISTORY
 * $Log$
 * Revision 1.2  2006/02/22  16:56:01  arthchan2003
 * Merged from SPHINX3_5_2_RCI_IRII_BRANCH: Added ms_mgau.[ch] into the trunk. It is a wrapper of ms_gauden and ms_senone
 * 
 * Revision 1.1.2.4  2005/09/25 18:55:19  arthchan2003
 * Added a flag to turn on and off precomputation.
 *
 * Revision 1.1.2.3  2005/08/03 18:53:44  dhdfu
 * Add memory deallocation functions.  Also move all the initialization
 * of ms_mgau_model_t into ms_mgau_init (duh!), which entails removing it
 * from decode_anytopo and friends.
 *
 * Revision 1.1.2.2  2005/08/02 21:05:38  arthchan2003
 * 1, Added dist and mgau_active as intermediate variable for computation. 2, Added ms_cont_mgau_frame_eval, which is a multi stream version of GMM computation mainly s3.0 family of tools. 3, Fixed dox-doc.
 *
 * Revision 1.1.2.1  2005/07/20 19:37:09  arthchan2003
 * Added a multi-stream cont_mgau (ms_mgau) which is a wrapper of both gauden and senone.  Add ms_mgau_init and model_set_mllr.  This allow eliminating 600 lines of code in decode_anytopo/align/allphone.
 *
 *
 *
 */

/* Local headers. */
#include "ms_mgau.h"

//#include <omp.h>

#include <pthread.h>

#define NTHREADS      8

pthread_spinlock_t spinlock;

struct timeval t1,t2;
float cuda_elapsedTime;
float cpu_elapsedTime;
float par_elapsedTime;

int i, start, tids[NTHREADS];
pthread_t threads[NTHREADS];
pthread_attr_t attr;

/*struct thread_param {
    gauden_t * g;
    int tid;
    int32 n_top;    
    mfcc_t** obs;
    gauden_dist_t ** out_dist;
};
struct thread_param *tp;*/

static gauden_t *global_g;
static int32 global_topn; 
static mfcc_t **global_obs;
static gauden_dist_t **global_dist;
static senone_t *global_sen;
static int16 *global_senscr;

static int32 global_best;

static int CPU = 1;

float calculateMiliseconds(struct timeval t1, struct timeval t2) {
	float elapsedTime;
	elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;
	elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;
	return elapsedTime;
}
          
static ps_mgaufuncs_t ms_mgau_funcs = {
    "ms",
    ms_cont_mgau_frame_eval, /* frame_eval */
    ms_mgau_mllr_transform,  /* transform */
    ms_mgau_free             /* free */
};

ps_mgau_t *
ms_mgau_init(acmod_t *acmod, logmath_t *lmath, bin_mdef_t *mdef)
{
    /* Codebooks */
    ms_mgau_model_t *msg;
    ps_mgau_t *mg;
    gauden_t *g;
    senone_t *s;
    cmd_ln_t *config;

    config = acmod->config;   
    
    msg = (ms_mgau_model_t *) ckd_calloc(1, sizeof(ms_mgau_model_t));
    msg->config = config;
    msg->g = NULL;
    msg->s = NULL;
    
    g = msg->g = gauden_init(cmd_ln_str_r(config, "-mean"),
                             cmd_ln_str_r(config, "-var"),
                             cmd_ln_float32_r(config, "-varfloor"),
                             lmath);

    /* Verify n_feat and veclen, against acmod. */
    if (g->n_feat != feat_dimension1(acmod->fcb)) {
        E_ERROR("Number of streams does not match: %d != %d\n",
                g->n_feat, feat_dimension1(acmod->fcb));
        goto error_out;
    }
    for (i = 0; i < g->n_feat; ++i) {
        if (g->featlen[i] != feat_dimension2(acmod->fcb, i)) {
            E_ERROR("Dimension of stream %d does not match: %d != %d\n", i,
                    g->featlen[i], feat_dimension2(acmod->fcb, i));
            goto error_out;
        }
    }

    s = msg->s = senone_init(msg->g,
                             cmd_ln_str_r(config, "-mixw"),
                             cmd_ln_str_r(config, "-senmgau"),
                             cmd_ln_float32_r(config, "-mixwfloor"),
                             lmath, mdef);

    s->aw = cmd_ln_int32_r(config, "-aw");

    /* Verify senone parameters against gauden parameters */
    if (s->n_feat != g->n_feat)
        E_FATAL("#Feature mismatch: gauden= %d, senone= %d\n", g->n_feat,
                s->n_feat);
    if (s->n_cw != g->n_density)
        E_FATAL("#Densities mismatch: gauden= %d, senone= %d\n",
                g->n_density, s->n_cw);
    if (s->n_gauden > g->n_mgau)
        E_FATAL("Senones need more codebooks (%d) than present (%d)\n",
                s->n_gauden, g->n_mgau);
    if (s->n_gauden < g->n_mgau)
        E_ERROR("Senones use fewer codebooks (%d) than present (%d)\n",
                s->n_gauden, g->n_mgau);

    msg->topn = cmd_ln_int32_r(config, "-topn");
    E_INFO("The value of topn: %d\n", msg->topn);
    if (msg->topn == 0 || msg->topn > msg->g->n_density) {
        E_WARN
            ("-topn argument (%d) invalid or > #density codewords (%d); set to latter\n",
             msg->topn, msg->g->n_density);
        msg->topn = msg->g->n_density;
    }

    msg->dist = (gauden_dist_t ***)
        ckd_calloc_3d(g->n_mgau, g->n_feat, msg->topn,
                      sizeof(gauden_dist_t));
    msg->mgau_active = (uint8 *) ckd_calloc(g->n_mgau, sizeof(int8));

    mg = (ps_mgau_t *)msg;
    mg->vt = &ms_mgau_funcs;
    
    pthread_attr_init(&attr);
    pthread_spin_init(&spinlock, 0);

    pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_JOINABLE);    
        
    return mg;
error_out:
    ms_mgau_free(ps_mgau_base(msg));
    return NULL;    
}

void
ms_mgau_free(ps_mgau_t * mg)
{
    ms_mgau_model_t *msg = (ms_mgau_model_t *)mg;
    if (msg == NULL)
        return;

    if (msg->g)
	gauden_free(msg->g);
    if (msg->s)
        senone_free(msg->s);
    if (msg->dist)
        ckd_free_3d((void *) msg->dist);
    if (msg->mgau_active)
        ckd_free(msg->mgau_active);
    
    ckd_free(msg);
    
    pthread_spin_destroy(&spinlock);
}

int
ms_mgau_mllr_transform(ps_mgau_t *s,
		       ps_mllr_t *mllr)
{
    ms_mgau_model_t *msg = (ms_mgau_model_t *)s;
    return gauden_mllr_transform(msg->g, mllr, msg->config);
}

//gauden_dist(gauden_t * g, int mgau, int32 n_top, mfcc_t** obs, gauden_dist_t ** out_dist)
// gauden_dist(g, gid, topn, feat, msg->dist[gid]);

__global__ void cuda_score(int32 n_mgau, int32 n_density, int32 featlen, mfcc_t * obs, mfcc_t ** mean, mfcc_t ** var, mfcc_t * det, gauden_dist_t * out_dist)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;  
    
    ms_mgau_model_t *msg = (ms_mgau_model_t *)mg;
        
    int32 temp = g->n_mgau;

            int32 temp = g->n_mgau;
            //#pragma parallel 
            //#pragma loop count min(1024)
            best = (int32) 0x7fffffff;
            //#pragma omp parallel for
           // #pragma parallel
            //#pragma acc kernels copyin(g,msg->dist) copy(senscr)
            //#pragma omp parallel for
            for (gid = 0; gid < temp; gid++) {
                                                             
                int mgau = gid;
                int32 i, j, d;
        
                //mfcc_t ** mean = g->mean[mgau][0];
                //mfcc_t ** var = g->var[mgau][0];
                //mfcc_t * det = g->det[mgau][0];
                //int32 n_density = g->n_density;               
                                     
                gauden_dist_t *worst;
                //gauden_dist_t * out_dist = msg->dist[gid][0];

                for (i = 0; i < topn; i++)
                    out_dist[i].dist = WORST_DIST;
                worst = &(out_dist[topn - 1]);

                for (d = 0; d < n_density; d++) {
                    mfcc_t *m;
                    mfcc_t *v;
                    mfcc_t dval;

                    m = mean[d];
                    v = var[d];
                    dval = det[d];

                    //for (i = 0; (i < featlen) && (dval >= worst->dist); i++) {
                    for (i = 0; (i < featlen); i++) {                        
                        mfcc_t diff;
                        diff = obs[i] - m[i];
                        /* The compiler really likes this to be a single
                         * expression, for whatever reason. */
                        dval -= diff * diff * v[i];
                    }

                    if ((i < featlen) || (dval < worst->dist))     /* Codeword d worse than worst */
                        continue;

                    /* Codeword d at least as good as worst so far; insert in the ordered list */
                    for (i = 0; (i < topn) && (dval < out_dist[i].dist); i++);
                    assert(i < topn);
                    for (j = topn - 1; j > i; --j)
                        out_dist[j] = out_dist[j - 1];
                    out_dist[i].dist = dval;
                    out_dist[i].id = d;
                }

                
                senscr[gid] = local_senone_eval(sen, gid, msg->dist[sen->mgau[gid]], topn);

           }       
}
            
void *
gauden_dist_thread(void *tid)
{
    // printf("%d\n", ((struct thread_param*)arg)->x);
    //    struct thread_param *tp = ((struct thread_param*)arg);
    
    	int k, start, *mytid, end;

        int iterations = global_g->n_mgau / NTHREADS;

	mytid = (int *) tid;
	start = (*mytid * iterations);
	end = start + iterations;
	//printf ("Thread %d doing iterations %d to %d\n",*mytid,start,end-1);       
                
        for (k=start; k < end ; k++) {           
	    //gauden_dist(global_g, k, global_topn, global_obs, global_dist[k]);
        }
}

void *
senone_eval_thread(void *tid)
{
       int32 local_best = (int32) 0x7fffffff;        
    
    // printf("%d\n", ((struct thread_param*)arg)->x);
    //    struct thread_param *tp = ((struct thread_param*)arg);
    
    	int k, start, *mytid, end;

        int iterations = global_sen->n_sen / NTHREADS;

	mytid = (int *) tid;
	start = (*mytid * iterations);
	end = start + iterations;
	//printf ("Thread %d doing iterations %d to %d\n",*mytid,start,end-1);      
        
	for (k = start; k < end; k++) {
	    //global_senscr[k] = senone_eval(global_sen, k, global_dist[global_sen->mgau[k]], global_topn);
	    if (local_best > global_senscr[k]) {
		local_best = global_senscr[k];
	    }
	}
        
        // update best local to global
        pthread_spin_lock(&spinlock);
        if (global_best > local_best) {
            global_best = local_best;
        }
        pthread_spin_unlock(&spinlock);
}

void *
dist_eval_thread(void *tid)
{
       int32 local_best = (int32) 0x7fffffff;        
    
    // printf("%d\n", ((struct thread_param*)arg)->x);
    //    struct thread_param *tp = ((struct thread_param*)arg);
    
    	int k, start, *mytid, end;

        int iterations = global_sen->n_sen / NTHREADS;

	mytid = (int *) tid;
	start = (*mytid * iterations);
	end = start + iterations;
	//printf ("Thread %d doing iterations %d to %d\n",*mytid,start,end-1);      
        
	for (k = start; k < end; k++) {
         //   gauden_dist(global_g, k, global_topn, global_obs, global_dist[k]);
	  //  global_senscr[k] = senone_eval(global_sen, k, global_dist[global_sen->mgau[k]], global_topn);
	    if (local_best > global_senscr[k]) {
		local_best = global_senscr[k];
	    }
	}
        
        // update best local to global
        pthread_spin_lock(&spinlock);
        if (global_best > local_best) {
            global_best = local_best;
        }
        pthread_spin_unlock(&spinlock);
}

/*
 * Compute senone score for one senone.
 * NOTE:  Remember that senone PDF tables contain SCALED, NEGATED logs3 values.
 * NOTE:  Remember also that PDF data may be transposed or not depending on s->n_gauden.
 */
__host__ __device__ int32
local_senone_eval(senone_t * s, int id, gauden_dist_t ** dist, int32 n_top)
{
    int32 scr;                  /* total senone score */
    int32 fden;                 /* Gaussian density */
    int32 fscr;                 /* senone score for one feature */
    int32 fwscr;                /* senone score for one feature, one codeword */
    int32 f, t;
    gauden_dist_t *fdist;

    assert((id >= 0) && (id < s->n_sen));
    assert((n_top > 0) && (n_top <= s->n_cw));

    scr = 0;

    for (f = 0; f < s->n_feat; f++) {
        int top;
        fdist = dist[f];

        /* Top codeword for feature f */
	top = fden = ((int32)fdist[0].dist + ((1<<SENSCR_SHIFT) - 1)) >> SENSCR_SHIFT;
        fscr = (s->n_gauden > 1)
	    ? (fden + -s->pdf[id][f][fdist[0].id])  /* untransposed */
	    : (fden + -s->pdf[f][fdist[0].id][id]); /* transposed */
        E_DEBUG(1, ("fden[%d][%d] l+= %d + %d = %d\n",
                    id, f, -(fscr - fden), -(fden-top), -(fscr-top)));
        /* Remaining of n_top codewords for feature f */
        for (t = 1; t < n_top; t++) {
	    fden = ((int32)fdist[t].dist + ((1<<SENSCR_SHIFT) - 1)) >> SENSCR_SHIFT;
            fwscr = (s->n_gauden > 1) ?
                (fden + -s->pdf[id][f][fdist[t].id]) :
                (fden + -s->pdf[f][fdist[t].id][id]);
            fscr = logmath_add(s->lmath, fscr, fwscr);
            E_DEBUG(1, ("fden[%d][%d] l+= %d + %d = %d\n",
                        id, f, -(fwscr - fden), -(fden-top), -(fscr-top)));
        }
	/* Senone scores are also scaled, negated logs3 values.  Hence
	 * we have to negate the stuff we calculated above. */
        scr -= fscr;
    }
    /* Downscale scores. */
    scr /= s->aw;

    /* Avoid overflowing int16 */
    if (scr > 32767)
      scr = 32767;
    if (scr < -32768)
      scr = -32768;
    return scr;
}

#define WORST_DIST	(int32)(0x80000000)

int32
ms_cont_mgau_frame_eval(ps_mgau_t * mg,
			int16 *senscr,
			uint8 *senone_active,
			int32 n_senone_active,
                        mfcc_t ** feat,
			int32 frame,
			int32 compallsen)
{
    ms_mgau_model_t *msg = (ms_mgau_model_t *)mg;
    int32 gid;
    int32 topn;
    int32 best;
    gauden_t *g;
    senone_t *sen;
    //int i;
    
    topn = ms_mgau_topn(msg);
    g = ms_mgau_gauden(msg);
    sen = ms_mgau_senone(msg);
    
    global_g = g;
    global_topn = topn;
    global_sen = sen;
   // global_dist = msg->dist;
    global_obs = feat;
    global_senscr = senscr;
    
    //printf("%d\n%d\n%d\n", g->n_mgau, msg->topn);
    
    // n_mgau = 5120
    //topn = 16             
                        
    if (compallsen) {
	int32 s;

        if (CPU) {

          //  gettimeofday(&t1, NULL);
            //#pragma omp parallel for 
            int32 temp = g->n_mgau;
            //#pragma parallel 
            //#pragma loop count min(1024)
            best = (int32) 0x7fffffff;
            //#pragma omp parallel for
           // #pragma parallel
            //#pragma acc kernels copyin(g,msg->dist) copy(senscr)
            //#pragma omp parallel for
            for (gid = 0; gid < temp; gid++) {
                                                  
                mfcc_t * obs = feat[0];                
                int32 featlen =  g->featlen[0];
                int mgau = gid;
                int32 i, j, d;
        
                mfcc_t ** mean = g->mean[mgau][0];
                mfcc_t ** var = g->var[mgau][0];
                mfcc_t * det = g->det[mgau][0];
                int32 n_density = g->n_density;               
                                     
                gauden_dist_t *worst;
                gauden_dist_t * out_dist = msg->dist[gid][0];

                for (i = 0; i < topn; i++)
                    out_dist[i].dist = WORST_DIST;
                worst = &(out_dist[topn - 1]);

                for (d = 0; d < n_density; d++) {
                    mfcc_t *m;
                    mfcc_t *v;
                    mfcc_t dval;

                    m = mean[d];
                    v = var[d];
                    dval = det[d];

                    //for (i = 0; (i < featlen) && (dval >= worst->dist); i++) {
                    for (i = 0; (i < featlen); i++) {                        
                        mfcc_t diff;
                        diff = obs[i] - m[i];
                        /* The compiler really likes this to be a single
                         * expression, for whatever reason. */
                        dval -= diff * diff * v[i];
                    }

                    if ((i < featlen) || (dval < worst->dist))     /* Codeword d worse than worst */
                        continue;

                    /* Codeword d at least as good as worst so far; insert in the ordered list */
                    for (i = 0; (i < topn) && (dval < out_dist[i].dist); i++);
                    assert(i < topn);
                    for (j = topn - 1; j > i; --j)
                        out_dist[j] = out_dist[j - 1];
                    out_dist[i].dist = dval;
                    out_dist[i].id = d;
                }

                
                senscr[gid] = local_senone_eval(sen, gid, msg->dist[sen->mgau[gid]], topn);

///                   local_senone_eval(senone_t * s, int id, gauden_dist_t ** dist, int32 n_top)


           }       
     
            //#pragma loop count min(1024)
            for (s = 0; s < sen->n_sen; s++) {
                //senscr[s] = senone_eval(sen, s, msg->dist[sen->mgau[s]], topn);
                if (best > senscr[s]) {
                    best = senscr[s];
                }
            }
           // gettimeofday(&t2, NULL);
          //  cpu_elapsedTime = calculateMiliseconds(t1, t2);
          //  printf("CPU SEQ Time=%4.3f ms\n",  cpu_elapsedTime);      

        } else {

            // GPU 
            global_best = (int32) 0x7fffffff;

            // PTHREAD        
          //  gettimeofday(&t1, NULL);
            /* for (i = 0; i < NTHREADS; i++) {
                tids[i] = i;
                pthread_create(&threads[i], &attr, dist_eval_thread, (void *) &tids[i]);
            }
            for (i = 0; i < NTHREADS; i++) {
                pthread_join(threads[i], NULL);
            }
            // compute gauden distance
           // for (i = 0; i < NTHREADS; i++) {
                tids[i] = i;
                pthread_create(&threads[i], &attr, gauden_dist_thread, (void *) &tids[i]);
            }        
            for (i = 0; i < NTHREADS; i++) {
                pthread_join(threads[i], NULL);
            }*/     
            // senone eval
            /*for (i = 0; i < NTHREADS; i++) {
                tids[i] = i;
                pthread_create(&threads[i], &attr, senone_eval_thread, (void *) &tids[i]);
            }        
            for (i = 0; i < NTHREADS; i++) {
                pthread_join(threads[i], NULL);
            }         */
            /*best = (int32) 0x7fffffff;
            for (s = 0; s < sen->n_sen; s++) {
                senscr[s] = senone_eval(sen, s, msg->dist[sen->mgau[s]], topn);
                if (best > senscr[s]) {
                    best = senscr[s];
                }
            }*/
          //  gettimeofday(&t2, NULL);

          //  par_elapsedTime = calculateMiliseconds(t1, t2);
            //printf("CPU PTHREAD Time=%4.3f ms\n",  par_elapsedTime);                       

           // if (global_best != best) {
             //   printf("ERROR computing best score!!  global_best: %d != best: %d", global_best, best);
          //  } else {
           //     best = global_best;
            //}

        }

        /// END OF PTHREAD
        
       // #pragma omp parallel for
	/* Normalize senone scores */
	for (s = 0; s < sen->n_sen; s++) {
	    int32 bs = senscr[s] - best;
	    if (bs > 32767)
		bs = 32767;
	    if (bs < -32768)
		bs = -32768;
	    senscr[s] = bs;
	}
    }
    else {
	int32 i, n;
	/* Flag all active mixture-gaussian codebooks */
	for (gid = 0; gid < g->n_mgau; gid++)
	    msg->mgau_active[gid] = 0;

	n = 0;
	for (i = 0; i < n_senone_active; i++) {
	    /* senone_active consists of deltas. */
	    int32 s = senone_active[i] + n;
	    msg->mgau_active[sen->mgau[s]] = 1;
	    n = s;
	}

	/* Compute topn gaussian density values (for active codebooks) */
	for (gid = 0; gid < g->n_mgau; gid++) {
	    if (msg->mgau_active[gid])
		gauden_dist(g, gid, topn, feat, msg->dist[gid]);
	}

	best = (int32) 0x7fffffff;
	n = 0;
	for (i = 0; i < n_senone_active; i++) {
	    int32 s = senone_active[i] + n;
	    senscr[s] = senone_eval(sen, s, msg->dist[sen->mgau[s]], topn);
	    if (best > senscr[s]) {
		best = senscr[s];
	    }
	    n = s;
	}

	/* Normalize senone scores */
	n = 0;
	for (i = 0; i < n_senone_active; i++) {
	    int32 s = senone_active[i] + n;
	    int32 bs = senscr[s] - best;
	    if (bs > 32767)
		bs = 32767;
	    if (bs < -32768)
		bs = -32768;
	    senscr[s] = bs;
	    n = s;
	}
    }

    return 0;
}
